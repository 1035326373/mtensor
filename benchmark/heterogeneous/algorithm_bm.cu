#include "hip/hip_runtime.h"
#include <bm_config.hpp>

#ifdef USE_CUDA

template <typename _ValueType>
__global__ void gold_fill_rank1_kernel(_ValueType *p_dst, int_t count, _ValueType v){
	for (int_t i = threadIdx.x + blockIdx.x * blockDim.x; i < count; i += blockDim.x * gridDim.x) {
		p_dst[i] = v;
	}
}

template <typename _ValueType>
void bm_gold_cu_fill_rank1(benchmark::State& state) {
	cuda::tensor<_ValueType, 1> ts_src(state.range(0));

	while (state.KeepRunning()) {
		cuda::parallel_execution_policy policy;
		policy.total_size(ts_src.size());
		cuda::configure_grid(policy, gold_fill_rank1_kernel<_ValueType>);
		gold_fill_rank1_kernel<<< policy.grid_size(),
			policy.block_size(),
			policy.shared_mem_bytes(),
			policy.stream() >>>(ts_src.data(), ts_src.size(), zero<_ValueType>::value());

		cuda::device_synchronize();

		benchmark::ClobberMemory();
	}

	auto bytes_size = static_cast<size_t>(ts_src.size()) * sizeof(_ValueType);
	state.SetBytesProcessed(state.iterations() * bytes_size);
}

#define BM_GOLD_CU_TENSOR_FILL_RANK1(ValueType) \
BENCHMARK_TEMPLATE1(bm_gold_cu_fill_rank1, byte)->RangeMultiplier(bm_config::range_multiplier<ValueType, 1, device_tag>())->Range(bm_config::min_shape<ValueType, 1, device_tag>(), bm_config::max_shape<ValueType, 1, device_tag>())->UseRealTime();

BM_GOLD_CU_TENSOR_FILL_RANK1(byte)
BM_GOLD_CU_TENSOR_FILL_RANK1(int16_t)
BM_GOLD_CU_TENSOR_FILL_RANK1(int32_t)
BM_GOLD_CU_TENSOR_FILL_RANK1(int64_t)
BM_GOLD_CU_TENSOR_FILL_RANK1(float)
BM_GOLD_CU_TENSOR_FILL_RANK1(double)
BM_GOLD_CU_TENSOR_FILL_RANK1(point3f)
BM_GOLD_CU_TENSOR_FILL_RANK1(point4f)

template <typename _ValueType>
__global__ void gold_copy_rank1_kernel(_ValueType *p_src, _ValueType *p_dst, int_t count){
	for (int_t i = threadIdx.x + blockIdx.x * blockDim.x; i < count; i += blockDim.x * gridDim.x) {
		p_dst[i] = p_src[i];
	}
}

template <typename _ValueType>
void bm_gold_cu_copy_rank1(benchmark::State& state) {
	cuda::tensor<_ValueType, 1> ts_src(state.range(0));
	cuda::tensor<_ValueType, 1> ts_dst(ts_src.size());
	fill(ts_src, zero<_ValueType>::value());

	while (state.KeepRunning()) {
		cuda::parallel_execution_policy policy;
		policy.total_size(ts_src.size());
		cuda::configure_grid(policy, gold_copy_rank1_kernel<_ValueType>);
		gold_copy_rank1_kernel<<< policy.grid_size(),
			policy.block_size(),
			policy.shared_mem_bytes(),
			policy.stream() >>>(ts_src.data(), ts_dst.data(), ts_src.size());

		cuda::device_synchronize();

		benchmark::ClobberMemory();
	}

	auto bytes_size = static_cast<size_t>(ts_src.size()) * sizeof(_ValueType);
	state.SetBytesProcessed(state.iterations() * bytes_size);
}

#define BM_GOLD_CU_TENSOR_COPY_RANK1(ValueType) \
BENCHMARK_TEMPLATE1(bm_gold_cu_copy_rank1, byte)->RangeMultiplier(bm_config::range_multiplier<ValueType, 1, device_tag>())->Range(bm_config::min_shape<ValueType, 1, device_tag>(), bm_config::max_shape<ValueType, 1, device_tag>())->UseRealTime();

BM_GOLD_CU_TENSOR_COPY_RANK1(byte)
BM_GOLD_CU_TENSOR_COPY_RANK1(int16_t)
BM_GOLD_CU_TENSOR_COPY_RANK1(int32_t)
BM_GOLD_CU_TENSOR_COPY_RANK1(int64_t)
BM_GOLD_CU_TENSOR_COPY_RANK1(float)
BM_GOLD_CU_TENSOR_COPY_RANK1(double)
BM_GOLD_CU_TENSOR_COPY_RANK1(point3f)
BM_GOLD_CU_TENSOR_COPY_RANK1(point4f)

#endif

#ifdef USE_HOST

template <typename _ValueType>
void bm_gold_host_fill_rank1(benchmark::State& state) {
	tensor<_ValueType, 1> ts_src(state.range(0));
	auto p_data = ts_src.data();
	auto size = ts_src.size();
	fill(ts_src, zero<_ValueType>::value());

	while (state.KeepRunning()) {
		for (int_t i = 0; i < size; ++i){
			p_data[i] = zero<_ValueType>::value();
		}

		benchmark::ClobberMemory();
	}

	auto bytes_size = static_cast<size_t>(ts_src.size()) * sizeof(_ValueType);
	state.SetBytesProcessed(state.iterations() * bytes_size);
}

#define BM_GOLD_HOST_TENSOR_FILL_RANK1(ValueType) \
BENCHMARK_TEMPLATE1(bm_gold_host_fill_rank1, byte)->RangeMultiplier(bm_config::range_multiplier<ValueType, 1, device_tag>())->Range(bm_config::min_shape<ValueType, 1, device_tag>(), bm_config::max_shape<ValueType, 1, device_tag>())->UseRealTime();

BM_GOLD_HOST_TENSOR_FILL_RANK1(byte)
BM_GOLD_HOST_TENSOR_FILL_RANK1(int16_t)
BM_GOLD_HOST_TENSOR_FILL_RANK1(int32_t)
BM_GOLD_HOST_TENSOR_FILL_RANK1(int64_t)
BM_GOLD_HOST_TENSOR_FILL_RANK1(float)
BM_GOLD_HOST_TENSOR_FILL_RANK1(double)
BM_GOLD_HOST_TENSOR_FILL_RANK1(point3f)
BM_GOLD_HOST_TENSOR_FILL_RANK1(point4f)

template <typename _ValueType>
void bm_gold_host_copy_rank1(benchmark::State& state) {
	tensor<_ValueType, 1> ts_src(state.range(0));
	tensor<_ValueType, 1> ts_dst(ts_src.size());
	fill(ts_src, zero<_ValueType>::value());
	auto p_src = ts_src.data();
	auto p_dst = ts_dst.data();
	auto size = ts_src.size();

	while (state.KeepRunning()) {
		for (int_t i = 0; i < size; ++i){
			p_dst[i] = p_src[i];
		}

		benchmark::ClobberMemory();
	}

	auto bytes_size = static_cast<size_t>(ts_src.size()) * sizeof(_ValueType);
	state.SetBytesProcessed(state.iterations() * bytes_size);
}

#define BM_GOLD_HOST_TENSOR_COPY_RANK1(ValueType) \
BENCHMARK_TEMPLATE1(bm_gold_host_copy_rank1, byte)->RangeMultiplier(bm_config::range_multiplier<ValueType, 1, device_tag>())->Range(bm_config::min_shape<ValueType, 1, device_tag>(), bm_config::max_shape<ValueType, 1, device_tag>())->UseRealTime();

BM_GOLD_HOST_TENSOR_COPY_RANK1(byte)
BM_GOLD_HOST_TENSOR_COPY_RANK1(int16_t)
BM_GOLD_HOST_TENSOR_COPY_RANK1(int32_t)
BM_GOLD_HOST_TENSOR_COPY_RANK1(int64_t)
BM_GOLD_HOST_TENSOR_COPY_RANK1(float)
BM_GOLD_HOST_TENSOR_COPY_RANK1(double)
BM_GOLD_HOST_TENSOR_COPY_RANK1(point3f)
BM_GOLD_HOST_TENSOR_COPY_RANK1(point4f)

#endif

template <typename _Tensor>
void bm_hete_tensor_fill(benchmark::State& state) {
	auto shape = pointi<_Tensor::rank>::all(state.range(0));
	_Tensor ts_src(shape);

	while (state.KeepRunning()) {
		fill(ts_src, zero<typename _Tensor::value_type>::value());
	#ifdef USE_CUDA
		cuda::device_synchronize();
	#endif

		benchmark::ClobberMemory();
	}

	auto bytes_size = static_cast<size_t>(ts_src.size()) * sizeof(decltype(ts_src[0]));
	state.SetBytesProcessed(state.iterations() * bytes_size);
}

#define BM_HETE_TENSOR_FILL(ValueType, Rank) \
auto bm_hete_tensor_fill_##ValueType##_rank##Rank = bm_hete_tensor_fill<HETE_TENSOR<ValueType, Rank>>; \
BENCHMARK(bm_hete_tensor_fill_##ValueType##_rank##Rank)->RangeMultiplier(bm_config::range_multiplier<ValueType, Rank, HETE_TAG>())->Range(bm_config::min_shape<ValueType, Rank, HETE_TAG>(), bm_config::max_shape<ValueType, Rank, HETE_TAG>())->UseRealTime();

#define BM_HETE_TENSOR_FILL_RANK1234(ValueType) \
BM_HETE_TENSOR_FILL(ValueType, 1) \
BM_HETE_TENSOR_FILL(ValueType, 2) \
BM_HETE_TENSOR_FILL(ValueType, 3) \
BM_HETE_TENSOR_FILL(ValueType, 4)

BM_HETE_TENSOR_FILL_RANK1234(byte)
BM_HETE_TENSOR_FILL_RANK1234(int16_t)
BM_HETE_TENSOR_FILL_RANK1234(int32_t)
BM_HETE_TENSOR_FILL_RANK1234(int64_t)
BM_HETE_TENSOR_FILL_RANK1234(float)
BM_HETE_TENSOR_FILL_RANK1234(double)
BM_HETE_TENSOR_FILL_RANK1234(point3f)
BM_HETE_TENSOR_FILL_RANK1234(point4f)

template <typename _Tensor>
void bm_hete_tensor_copy(benchmark::State& state) {
	auto shape = pointi<_Tensor::rank>::all(state.range(0));
	_Tensor ts_src(shape);
	_Tensor ts_dst(shape);
	fill(ts_src, zero<typename _Tensor::value_type>::value());

	while (state.KeepRunning()) {
		copy(ts_src, ts_dst);
	#ifdef USE_CUDA
		cuda::device_synchronize();
	#endif

		benchmark::ClobberMemory();
	}

	auto bytes_size = static_cast<size_t>(ts_src.size()) * sizeof(decltype(ts_src[0]));
	state.SetBytesProcessed(state.iterations() * bytes_size);
}

#define BM_HETE_TENSOR_COPY(ValueType, Rank) \
auto bm_hete_tensor_copy_##ValueType##_rank##Rank = bm_hete_tensor_copy<HETE_TENSOR<ValueType, Rank>>; \
BENCHMARK(bm_hete_tensor_copy_##ValueType##_rank##Rank)->RangeMultiplier(bm_config::range_multiplier<ValueType, Rank, HETE_TAG>())->Range(bm_config::min_shape<ValueType, Rank, HETE_TAG>(), bm_config::max_shape<ValueType, Rank, HETE_TAG>())->UseRealTime();

#define BM_HETE_TENSOR_COPY_RANK1234(ValueType) \
BM_HETE_TENSOR_COPY(ValueType, 1) \
BM_HETE_TENSOR_COPY(ValueType, 2) \
BM_HETE_TENSOR_COPY(ValueType, 3) \
BM_HETE_TENSOR_COPY(ValueType, 4)

BM_HETE_TENSOR_COPY_RANK1234(byte)
BM_HETE_TENSOR_COPY_RANK1234(int16_t)
BM_HETE_TENSOR_COPY_RANK1234(int32_t)
BM_HETE_TENSOR_COPY_RANK1234(int64_t)
BM_HETE_TENSOR_COPY_RANK1234(float)
BM_HETE_TENSOR_COPY_RANK1234(double)
BM_HETE_TENSOR_COPY_RANK1234(point3f)
BM_HETE_TENSOR_COPY_RANK1234(point4f)

//
