﻿#include <benchmark/benchmark.h>
#include <matazure/bm_config.hpp>
#include <mtensor.hpp>

int main(int argc, char** argv) {
#ifdef USE_CUDA
	matazure::cuda::set_device(0);
#endif
	::benchmark::Initialize(&argc, argv);
	::benchmark::RunSpecifiedBenchmarks();
}
