#include "hip/hip_runtime.h"
#include "../bm_config.hpp"

void bm_cuda_tensor_matrix_mul(benchmark::State& state) {
    const int BLOCK_SIZE = 16;
    typedef dim<BLOCK_SIZE, BLOCK_SIZE> BLOCK_DIM;
    pointi<2> block_dim = BLOCK_DIM::value();
    pointi<2> grid_dim;
    fill(grid_dim, state.range(0));
    pointi<2> global_dim = block_dim * grid_dim;

    int M = global_dim[0];
    int N = global_dim[1];
    int K = BLOCK_SIZE * 100;

    cuda::tensor<float, 2> cmat_a(pointi<2>{M, K});
    cuda::tensor<float, 2> cmat_b(pointi<2>{K, N});
    cuda::tensor<float, 2> cmat_c(pointi<2>{M, N});

    while (state.KeepRunning()) {
        cuda::block_for_index<BLOCK_DIM>(grid_dim,
                                         [=] __device__(cuda::block_index<BLOCK_DIM> block_idx) {
                                             auto row = block_idx.local[0];
                                             auto col = block_idx.local[1];
                                             auto global_row = block_idx.global[0];
                                             auto global_col = block_idx.global[1];

                                             __shared__ local_tensor<float, BLOCK_DIM> local_a;
                                             __shared__ local_tensor<float, BLOCK_DIM> local_b;

                                             float sum = 0.0f;
                                             for (int_t i = 0; i < K; i += BLOCK_SIZE) {
                                                 local_a(row, col) = cmat_a(global_row, col + i);
                                                 local_b(row, col) = cmat_b(row + i, global_col);
                                                 cuda::syncthreads();

#pragma unroll
                                                 for (int_t N = 0; N < BLOCK_SIZE; N++) {
                                                     sum += local_a(row, N) * local_b(N, col);
                                                 }
                                             }

                                             cmat_c(block_idx.global) += sum;
                                         });
    }

    state.SetItemsProcessed(state.iterations() * M * N * K);
}

BENCHMARK(bm_cuda_tensor_matrix_mul)->Arg(1_K);