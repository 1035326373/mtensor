
#include "../bm_algorithm.hpp"

void bm_cuda_cudaMemcpy(benchmark::State& state) {
    int ts_size = state.range(0);
    cuda::tensor<float, 1> cts_src(ts_size);
    cuda::tensor<float, 1> cts_dst(ts_size);

    while (state.KeepRunning()) {
        hipMemcpy(cts_dst.data(), cts_src.data(), sizeof(cts_src[0]) * cts_src.size(),
                   hipMemcpyDefault);
        hipDeviceSynchronize();
    }

    state.SetBytesProcessed(state.iterations() * static_cast<size_t>(cts_src.size()) *
                            sizeof(float));
}

auto bm_cuda_tensor1f_copy = bm_tensor_copy<cuda::tensor<float, 1>>;
auto bm_cuda_tensor2f_copy = bm_tensor_copy<cuda::tensor<float, 2>>;
BENCHMARK(bm_cuda_tensor1f_copy)->Arg(1_G);
BENCHMARK(bm_cuda_tensor2f_copy)->Arg(32_K);

BENCHMARK_MAIN();
