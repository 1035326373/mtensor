#include "hip/hip_runtime.h"
#include "../bm_config.hpp"

__global__ void kernel_freq(float* p_src, float* p_dst, int_t size) {
    for (int_t i = threadIdx.x + blockIdx.x * blockDim.x; i < size; i += blockDim.x * gridDim.x) {
        auto tmp = p_dst[i];
        for (int_t k = 0; k < 1000000; ++k) {
            tmp *= 1.01f;
        }
        p_src[i] = tmp;
    }
}

void bm_cuda_lauch_kernel_freq(benchmark::State& state) {
    int ts_size = state.range(0);
    cuda::tensor<float, 1> ts_src(ts_size);
    cuda::tensor<float, 1> ts_dst(ts_size);

    cuda::parallel_execution_policy policy;
    policy.total_size(ts_src.size());
    cuda::configure_grid(policy, kernel_freq);

    while (state.KeepRunning()) {
        kernel_freq<<<policy.grid_dim(), policy.block_dim(), policy.shared_mem_bytes(),
                      policy.stream()>>>(ts_src.data(), ts_dst.data(), ts_src.size());
        hipDeviceSynchronize();
    }

    state.SetItemsProcessed(state.iterations() * static_cast<size_t>(ts_src.size()) * 1000000);
}

BENCHMARK(bm_cuda_lauch_kernel_freq)->Arg(1_M);
