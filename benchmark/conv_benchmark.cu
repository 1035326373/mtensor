
#include <hip/hip_runtime.h>
﻿//#include <benchmark/benchmark.h>
//#include <matazure/tensor>
//#include <matazure/cuda/puzzle/conv.hpp>
//
//using namespace matazure;
//
//__constant__ static_tensor<float, 3, 3> mask;
//
//template <typename _ValueType>
//void BM_cu_conv_global(benchmark::State& state) {
//	pointi<2> ext;
//	fill(ext, state.range(0));
//	cu_tensor<_ValueType, 2> ts_src(ext);
//	fill(mask, _ValueType(0));
//
//	while (state.KeepRunning()) {
//		auto ts_re = cuda::puzzle::conv<8,8>(ts_src, mask);
//	}
//
//	auto bytes_size = static_cast<size_t>(ts_src.size()) * sizeof(_ValueType);
//	state.SetBytesProcessed(state.iterations() * bytes_size);
//}
//
//
//BENCHMARK_TEMPLATE1(BM_cu_conv_global, float)->RangeMultiplier(2)->Range(128, 4096)->UseRealTime();
