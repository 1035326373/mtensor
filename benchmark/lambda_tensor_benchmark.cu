﻿#include <benchmark/benchmark.h>
#include <matazure/tensor>

using namespace matazure;

static void BM_linear_lambda_tensor_persist_gold(benchmark::State &st) {
	tensor<float, 1> tsf1(st.range(0));
	while (st.KeepRunning()) {
		tensor<float, 1> ts_re(tsf1.extent());
		for (int_t i = 0; i < ts_re.size(); ++i) {
			ts_re[i] = 2.0f * tsf1[i];
		}
	}

	auto bytes_size = static_cast<size_t>(tsf1.size()) * sizeof(decltype(tsf1[0]));
	st.SetBytesProcessed(st.iterations() * bytes_size);
}

static void BM_linear_lambda_tensor_persist(benchmark::State &st) {
	tensor<float, 1> tsf1(st.range(0));
	while (st.KeepRunning()) {
		auto tsf1_re = make_lambda(tsf1.extent(), [tsf1](int_t i) {
			return 2.0f * tsf1[i];
		}).persist();
	}

	auto bytes_size = static_cast<size_t>(tsf1.size()) * sizeof(decltype(tsf1[0]));
	st.SetBytesProcessed(st.iterations() * bytes_size);
}

BENCHMARK(BM_linear_lambda_tensor_persist_gold)->Range(1 << 10, 1 << 28)->UseRealTime();
BENCHMARK(BM_linear_lambda_tensor_persist)->Range(1 << 10, 1 << 28)->UseRealTime();
