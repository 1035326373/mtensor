﻿#include <matazure/tensor>

using namespace matazure;

__constant__ static_tensor<float, dim< 3, 3>> mask;
//之所以使用宏来定义卷积函数，是因为cuda的constant内存必须全局申明
//声明一个叫做conv_global的卷积函数 用mask作为卷积核
MATAZURE_CUDA_PUZZEL_CONV_GLOBAL(conv_global, mask)
//分块的卷积实现
MATAZURE_CUDA_PUZZEL_CONV_BLOCK_ALIGNED(conv_block, mask)
//边缘不处理的卷积实现
MATAZURE_CUDA_PUZZEL_CONV_BLOCK_CRACK(conv_block_crack, mask)
//处理了边缘的卷积实现
MATAZURE_CUDA_PUZZEL_CONV_BLOCK_OVERLAP(conv_block_overlap, mask)

int main() {
	static_tensor<float, dim< 3, 3>> host_mask;
	fill(host_mask, 1.0f / host_mask.size());
	cuda::copy_symbol(host_mask, mask);

	tensor<byte, 2> gray(512, 512);
	io::read_raw_data("data/lena_gray8_512x512.raw_data", gray);
	auto cu_gray = mem_clone(gray, device_t{});

	auto lcts_conv = cuda::puzzle::conv_global(tensor_cast<float>(clamp_zero(cu_gray)));
	auto cts_conv = apply(lcts_conv, unary::saturate_convert<byte>{}).persist();
	cuda::device_synchronize();
	auto ts_conv = mem_clone(cts_conv, host_t{});
	io::write_raw_data("data/lena_gray8_conv_512x512.raw_data", ts_conv);

	cuda::tensor<float, 2> cts_conv_block(cu_gray.shape());
	cuda::puzzle::conv_block<dim<16, 16>>(tensor_cast<float>(cu_gray), cts_conv_block);
	auto cts_byte_conv_block = apply(cts_conv_block, unary::saturate_convert<byte>{}).persist();
	cuda::device_synchronize();
	auto ts_byte_conv_block = mem_clone(cts_byte_conv_block, host_t{});
	io::write_raw_data("data/lena_gray8_conv_block_512x512.raw_data", ts_byte_conv_block);

	cuda::tensor<float, 2> cts_conv_block_crack(cu_gray.shape());
	cuda::puzzle::conv_block_crack<dim<32, 32>>(tensor_cast<float>(clamp_zero(cu_gray)), cts_conv_block_crack);
	auto cts_byte_conv_block_crack = apply(cts_conv_block_crack, unary::saturate_convert<byte>{}).persist();
	cuda::device_synchronize();
	auto ts_byte_conv_block_crack = mem_clone(cts_byte_conv_block_crack, host_t{});
	io::write_raw_data("data/lena_gray8_conv_block_crack_512x512.raw_data", ts_byte_conv_block_crack);

	cuda::tensor<float, 2> cts_conv_block_overlap(cu_gray.shape());
	cuda::puzzle::conv_block_overlap<dim<16, 16>>(tensor_cast<float>(clamp_zero(cu_gray)), cts_conv_block_overlap);
	auto cts_byte_conv_block_overlap = apply(cts_conv_block_overlap, unary::saturate_convert<byte>{}).persist();
	cuda::device_synchronize();
	auto ts_byte_conv_block_overlap = mem_clone(cts_byte_conv_block_overlap, host_t{});
	io::write_raw_data("data/lena_gray8_conv_block_overlap_512x512.raw_data", ts_byte_conv_block_overlap);

	return 0;
}
