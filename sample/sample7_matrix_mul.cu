#include "hip/hip_runtime.h"
#include <matazure/tensor>

using namespace matazure;

int main(int argc, char * argv[]) {
	const int BLOCK_SIZE = 16;
	typedef dim<BLOCK_SIZE, BLOCK_SIZE> BLOCK_DIM;
	pointi<2> block_dim = BLOCK_DIM::value();
	pointi<2> grid_dim{8, 8};
	pointi<2> global_dim = block_dim * grid_dim;
	int M = global_dim[0];
	int N = global_dim[1];
	int K = BLOCK_SIZE * 4;

	cuda::tensor<float, 2> cmat_a(pointi<2>{M, K});
	cuda::tensor<float, 2> cmat_b(pointi<2>{K, N});
	cuda::tensor<float, 2> cmat_c(pointi<2>{M, N});

	cuda::fill(cmat_a, 1.0f);
	cuda::fill(cmat_b, 2.0f);
	cuda::device_synchronize();

	cuda::block_for_index<BLOCK_DIM>(grid_dim, [=] __device__ (cuda::block_index<BLOCK_DIM> block_idx) {
		auto row = block_idx.local[0];
		auto col = block_idx.local[1];
		auto global_row = block_idx.global[0];
		auto global_col = block_idx.global[1];

		__shared__ static_tensor<float, BLOCK_DIM> local_a;
		__shared__ static_tensor<float, BLOCK_DIM> local_b;

		float sum = 0.0f;
		for (int_t i = 0; i < K; i += BLOCK_SIZE) {
			local_a(row, col) = cmat_a(global_row, col + i);
			local_b(row, col) = cmat_b(row + i, global_col);
			cuda::sync_threads();

			for (int_t N = 0; N < BLOCK_SIZE; N++) {
				sum += local_a(row, N) * local_b(N, col);
			}
		}

		cmat_c(block_idx.global) = sum;
	});
	cuda::device_synchronize();

	return 0;
}
