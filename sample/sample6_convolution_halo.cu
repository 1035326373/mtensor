#include "hip/hip_runtime.h"
#include <mtensor.hpp>
#include "image_helper.hpp"

using namespace matazure;

int main(int argc, char* argv[]) {
    if (argc < 2) {
        std::cout << "sample6_convolution input_image" << std::endl;
        return -1;
    }

    //读取图像
    tensor<pointb<3>, 2> img_rgb = read_rgb_image(argv[1]);
    //使用均值滤波器
    tensor<pointf<3>, 2> kernel_mean(pointi<2>{3, 3});
    fill(kernel_mean, pointf<3>{0.111f, 0.111f, 0.111f});

    //向GPU拷贝数据
    cuda::tensor<pointb<3>, 2> cimg_rgb(img_rgb.shape());
    mem_copy(img_rgb, cimg_rgb);
    cuda::tensor<pointf<3>, 2> ckernel_mean(kernel_mean.shape());
    mem_copy(kernel_mean, ckernel_mean);

    //结果图像
    cuda::tensor<pointf<3>, 2> cimg_mean(img_rgb.shape());

    typedef dim<16, 16> BLOCK_DIM;
    pointi<2> block_dim = BLOCK_DIM::value();
    auto grid_dim = (img_rgb.shape() + block_dim - pointi<2>{1, 1}) / block_dim;
    auto padding = kernel_mean.shape() / 2;

    cuda::block_for_index<BLOCK_DIM>(grid_dim, [=] __device__(
                                                   cuda::block_index<BLOCK_DIM> block_idx) {
        //使用shared memory以获取更好的速度
        __shared__ local_tensor<pointf<3>, BLOCK_DIM> sh_ts_block;
        //若是无效区域则填充0
        if (inside_rect(block_idx.global, pointi<2>{0, 0}, cimg_rgb.shape())) {
            sh_ts_block(block_idx.local) = point_cast<float>(cimg_rgb(block_idx.global));
        } else {
            sh_ts_block(block_idx.local) = pointf<3>{0, 0, 0};
        }

        cuda::syncthreads();

        if (inside_rect(block_idx.local, padding, block_idx.block_dim - ckernel_mean.shape() + 1) &&
            inside_rect(block_idx.global, pointi<2>{0, 0}, cimg_rgb.shape())) {
            auto sum = pointf<3>{0, 0, 0};
            //在__device__ lambda算子里，一样可以使用matazure::for_index操作
            for_index(pointi<2>{0, 0}, ckernel_mean.shape(), [&](const pointi<2>& idx) {
                sum += sh_ts_block(block_idx.local + idx - padding) * ckernel_mean(idx);
            });
            cimg_mean[block_idx.global] = sum;
        }
    });

    //转换float类型到byte类型
    cuda::tensor<pointb<3>, 2> cimg_mean_byte(cimg_mean.shape());
    cuda::transform(cimg_mean, cimg_mean_byte,
                    [] __device__(pointf<3> pixel) { return point_cast<byte>(pixel); });
    //向主机写入图像
    tensor<pointb<3>, 2> img_mean(cimg_mean_byte.shape());
    mem_copy(cimg_mean_byte, img_mean);
    write_rgb_png("mean_" + std::string(argv[1]), img_mean);

    return 0;
}
