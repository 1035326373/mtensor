﻿#include <iostream>
#include <mtensor.hpp>

using namespace matazure;

int main(int argc, char* argv[]) {
    pointi<2> shape{5, 5};
    tensor<float, 2> ts_a(shape);
    tensor<float, 2> ts_b(shape);
    tensor<float, 2> ts_c(shape);
    fill(ts_a, 1.0f);
    fill(ts_b, 2.0f);

    //构造gpu上的tensor
    cuda::tensor<float, 2> cts_a(shape);
    cuda::tensor<float, 2> cts_b(shape);
    cuda::tensor<float, 2> cts_c(shape);

    //将cpu上的数据拷贝到gpu上
    mem_copy(ts_a, cts_a);
    mem_copy(ts_b, cts_b);

    //使用cuda  lambda算子 需要申明__device__ __host__
    auto functor = [cts_a, cts_b, cts_c] __device__ __host__(pointi<2> index) {
        cts_c(index) = cts_a(index) + cts_b(index);
    };

    cuda::for_index(shape, functor);

    //将gpu上数据拷贝会cpu
    mem_copy(cts_c, ts_c);

    //打印输出
    for_each(ts_c, [](float e) { std::cout << e << ", "; });
    std::cout << std::endl;

    return 0;
}
